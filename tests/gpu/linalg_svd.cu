#include "hip/hip_runtime.h"
#include "../catch.hpp"

#include <fml/_internals/arraytools/src/arraytools.hpp>
#include <fml/gpu/gpumat.hh>
#include <fml/gpu/gpuvec.hh>
#include <fml/gpu/linalg.hh>

using namespace arraytools;

extern fml::card_sp_t c;


TEMPLATE_TEST_CASE("svd - values - ts", "[linalg]", float, double)
{
  len_t m = 3;
  len_t n = 2;
  
  fml::gpuvec<TestType> v(c, n);
  v.set(0, 2);
  v.set(1, 5);
  
  fml::gpumat<TestType> x(c, m, n);
  
  SECTION("qrsvd")
  {
    x.fill_diag(v);
    fml::gpuvec<TestType> s_ts(c);
    fml::linalg::qrsvd(x, s_ts);
    s_ts.rev();
    REQUIRE( v == s_ts );
  }
  
  SECTION("cpsvd")
  {
    x.fill_diag(v);
    fml::gpuvec<TestType> s_cp(c);
    fml::linalg::cpsvd(x, s_cp);
    
    s_cp.rev();
    REQUIRE( v == s_cp );
  }
}



TEMPLATE_TEST_CASE("svd - ts", "[linalg]", float, double)
{
  // matrix from https://en.wikipedia.org/wiki/Singular_value_decomposition#Example
  fml::gpumat<TestType> x(c, 5, 4);
  x.fill_zero();
  x.set(0, 0, 1);
  x.set(4, 0, 2);
  x.set(1, 3, 2);
  x.set(2, 1, 3);
  
  fml::gpuvec<TestType> s(c);
  fml::gpumat<TestType> u(c), vt(c);
  
  SECTION("qrsvd")
  {
    fml::linalg::qrsvd(x, s, u, vt);
    
    REQUIRE( fltcmp::eq(s.get(0), 3) );
    REQUIRE( fltcmp::eq(s.get(1), sqrt(5)) );
    REQUIRE( fltcmp::eq(s.get(2), 2) );
    
    REQUIRE( fltcmp::eq(fabs(u.get(0, 1)), sqrtf(0.2f)) );
    REQUIRE( fltcmp::eq(fabs(u.get(4, 1)), sqrtf(0.8f)) );
    
    REQUIRE( fltcmp::eq(fabs(vt.get(1, 0)), 1) );
    REQUIRE( fltcmp::eq(fabs(vt.get(0, 1)), 1) );
  }
  
  SECTION("cpsvd")
  {
    fml::linalg::cpsvd(x, s, u, vt);
    
    REQUIRE( fltcmp::eq(s.get(0), 3) );
    REQUIRE( fltcmp::eq(s.get(1), sqrt(5)) );
    REQUIRE( fltcmp::eq(s.get(2), 2) );
    
    REQUIRE( fltcmp::eq(fabs(u.get(0, 1)), sqrtf(0.2f)) );
    REQUIRE( fltcmp::eq(fabs(u.get(4, 1)), sqrtf(0.8f)) );
    
    REQUIRE( fltcmp::eq(fabs(vt.get(1, 0)), 1) );
    REQUIRE( fltcmp::eq(fabs(vt.get(0, 1)), 1) );
  }
}



TEMPLATE_TEST_CASE("svd - values - sf", "[linalg]", float, double)
{
  len_t m = 2;
  len_t n = 3;
  
  fml::gpuvec<TestType> v(c, m);
  v.set(0, 2);
  v.set(1, 5);
  
  fml::gpumat<TestType> x(c, m, n);
  
  SECTION("qrsvd")
  {
    x.fill_diag(v);
    fml::gpuvec<TestType> s_ts(c);
    fml::linalg::qrsvd(x, s_ts);
    s_ts.rev();
    REQUIRE( v == s_ts );
  }
  
  SECTION("cpsvd")
  {
    x.fill_diag(v);
    fml::gpuvec<TestType> s_cp(c);
    fml::linalg::cpsvd(x, s_cp);
    
    s_cp.rev();
    REQUIRE( v == s_cp );
  }
}



TEMPLATE_TEST_CASE("svd - sf", "[linalg]", float, double)
{
  // matrix from https://en.wikipedia.org/wiki/Singular_value_decomposition#Example
  fml::gpumat<TestType> x(c, 4, 5);
  x.fill_zero();
  x.set(0, 0, 1);
  x.set(3, 1, 2);
  x.set(1, 2, 3);
  x.set(0, 4, 2);
  
  fml::gpuvec<TestType> s(c);
  fml::gpumat<TestType> u(c), vt(c);
  
  SECTION("qrsvd")
  {
    fml::linalg::qrsvd(x, s, u, vt);
    
    REQUIRE( fltcmp::eq(s.get(0), 3) );
    REQUIRE( fltcmp::eq(s.get(1), sqrt(5)) );
    REQUIRE( fltcmp::eq(s.get(2), 2) );
    
    REQUIRE( fltcmp::eq(fabs(u.get(1, 0)), 1) );
    REQUIRE( fltcmp::eq(fabs(u.get(0, 1)), 1) );
    
    REQUIRE( fltcmp::eq(fabs(vt.get(1, 0)), sqrtf(0.2f)) );
    REQUIRE( fltcmp::eq(fabs(vt.get(1, 4)), sqrtf(0.8f)) );
  }
  
  SECTION("cpsvd")
  {
    fml::linalg::cpsvd(x, s, u, vt);
    
    REQUIRE( fltcmp::eq(s.get(0), 3) );
    REQUIRE( fltcmp::eq(s.get(1), sqrt(5)) );
    REQUIRE( fltcmp::eq(s.get(2), 2) );
    
    REQUIRE( fltcmp::eq(fabs(u.get(1, 0)), 1) );
    REQUIRE( fltcmp::eq(fabs(u.get(0, 1)), 1) );
    
    REQUIRE( fltcmp::eq(fabs(vt.get(1, 0)), sqrtf(0.2f)) );
    REQUIRE( fltcmp::eq(fabs(vt.get(1, 4)), sqrtf(0.8f)) );
  }
}
