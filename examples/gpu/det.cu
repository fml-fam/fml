#include "hip/hip_runtime.h"
#include <fml/gpu.hh>


static inline void print_det(int sign, float modulus)
{
  printf("sgn = %d\n", sign);
  printf("mod = %f\n", modulus);
  printf("sgn*exp(mod) = %f\n", sign*exp(modulus));
  printf("\n");
}



int main()
{
  auto c = fml::new_card(0);
  
  len_t n = 2;
  
  fml::gpumat<float> x(c, n, n);
  x.fill_linspace(1, n*n);
  
  x.info();
  x.print(0);
  
  int sign;
  float modulus;
  
  fml::linalg::det(x, sign, modulus);
  print_det(sign, modulus);
  
  n = 4;
  x.resize(n, n);
  x.fill_linspace(1, n*n);
  
  x.info();
  x.print(0);
  
  fml::linalg::det(x, sign, modulus);
  print_det(sign, modulus);
  
  return 0;
}
