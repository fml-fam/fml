#include "hip/hip_runtime.h"
#include <gpu/card.hh>
#include <gpu/gpuhelpers.hh>
#include <gpu/gpumat.hh>
#include <gpu/linalg.hh>


int main()
{
  auto c = gpuhelpers::new_card(0);
  c->info();
  
  len_t n = 5;
  gpumat<float> x(c, n, n);
  x.fill_linspace(1.f, (float) n*n);
  
  gpumat<float> y(c, n, n);
  y.fill_linspace(1.f, (float) n*n);
  
  gpumat<__half> xh(c), yh(c);
  gpuhelpers::gpu2gpu(x, xh);
  gpuhelpers::gpu2gpu(y, yh);
  
  gpumat<__half> zh = linalg::matmult(false, false, (__half)1.f, xh, yh);
  zh.info();
  zh.print(0);
  
  gpumat<float> z(c);
  gpuhelpers::gpu2gpu(zh, z);
  z.info();
  z.print(0);
  
  return 0;
}
