#include "hip/hip_runtime.h"
#include <fml/gpu.hh>


int main()
{
  auto c = fml::new_card(0);
  c->info();
  
  len_t n = 5;
  fml::gpumat<float> x(c, n, n);
  x.fill_linspace(1.f, (float) n*n);
  
  fml::gpumat<float> y(c, n, n);
  y.fill_linspace(1.f, (float) n*n);
  
  fml::gpumat<__half> xh(c), yh(c);
  fml::copy::gpu2gpu(x, xh);
  fml::copy::gpu2gpu(y, yh);
  
  fml::gpumat<__half> zh = fml::linalg::matmult(false, false, (__half)1.f, xh, yh);
  zh.info();
  zh.print(0);
  
  fml::gpumat<float> z(c);
  fml::copy::gpu2gpu(zh, z);
  z.info();
  z.print(0);
  
  return 0;
}
